#include "hip/hip_runtime.h"

/*
A simple example demonstrating how to write a cuda kernel
The kernel adds 2 arrays and saves the output to a 3rd array

First, we show the output using a serial addition through serial_add_arrays

Then, we show the same output using the function parallel_add_arrays which is a stub function
that invokes vecAddKernel. Inside parallel_add_arrays, first we allocate the required memory
in host (GPU) for the 3 arrays. We can invoke the kernel with a block_size of 32. In other words,
we request 32 thread for each block. Finally, we copy the result in 3rd array back from
device (GPU) to host (CPU) memory and free up the host memory.

Commands to run on a Linux machine and the output:
nvcc -c vector_ad.cu
nvcc vector_add.o -o vector_add
./vector_add

Output of serial add: 4, 6, 4, 11, 14, 
Output of parallel add: 4, 6, 4, 11, 14,
*/

#include<hip/hip_runtime.h>
#include<stdio.h>

__global__
void vecAddKernel(float *a, float *b, float *c, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n)
        c[i] = a[i] + b[i];
}

void parallel_add_arrays(float *a, float *b, float *c, int n){
    int size = n * sizeof(float);
    float *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, size);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_b, size);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_c, size);

    int block_size = 32;
    vecAddKernel<<<ceil(n/(float)block_size), block_size>>>(d_a, d_b, d_c, n);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

void serial_add_arrays(float *a, float *b, float *c, int n){
    for(int i = 0; i < n; i++)
        c[i] = a[i] + b[i];
}

void print_arr(float *a, int n, char *header){
    std::cout<<header;
    for(int i = 0; i < n; i++)
        std::cout<<a[i]<<", ";
    std::cout<<std::endl;
}

int main(){
    float A[5] = {1, 2, 3, 4, 5};
    float B[5] = {3, 4, 1, 7, 9};
    float C[5];
    int array_size = 5;

    serial_add_arrays(A, B, C, array_size);
    print_arr(C, array_size, "Output of serial add: ");
    parallel_add_arrays(A, B, C, array_size);
    print_arr(C, array_size, "Output of parallel add: ");

    return 0;
};